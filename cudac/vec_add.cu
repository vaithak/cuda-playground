
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vec_add(float *a, float *b, float *res, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        res[i] = a[i] + b[i];
    }
}

int main() {
    int n = 100;

    // allocate memory on the host.
    float *a, *b, *res;
    a = (float *)malloc(n * sizeof(float));
    b = (float *)malloc(n * sizeof(float));
    res = (float *)malloc(n * sizeof(float));

    // initialize the host arrays.
    for (int i = 0; i < n; i++) {
        a[i] = i;
        b[i] = i;
    }

    // allocate memory on the device.
    float *d_a, *d_b, *d_res;
    hipMalloc((void**)&d_a, n * sizeof(float));
    hipMalloc((void**)&d_b, n * sizeof(float));
    hipMalloc((void**)&d_res, n * sizeof(float));

    // copy the host arrays to the device.
    hipMemcpy((void*)d_a, (void*)a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy((void*)d_b, (void*)b, n * sizeof(float), hipMemcpyHostToDevice);

    // launch the kernel.
    vec_add<<<(n + 255) / 256, 256>>>(d_a, d_b, d_res, n);

    // copy the result back to the host.
    hipMemcpy((void*)res, (void*)d_res, n * sizeof(float), hipMemcpyDeviceToHost);

    // print the first and last 10 elements of the result.
    printf("First 10 elements:\n");
    for (int i = 0; i < 10; i++) {
        printf("\t%f + %f = %f\n", a[i], b[i], res[i]);
    }
    printf("...\n");
    printf("Last 10 elements:\n");
    for (int i = 0; i < 10; i++) {
        printf("\t%f + %f = %f\n", a[n-10+i], b[n-10+i], res[n-10+i]);
    }

    // free the memory.
    free(a);
    free(b);
    free(res);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_res);

    return 0;
}
